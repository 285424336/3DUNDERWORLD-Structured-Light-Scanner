#include "hip/hip_runtime.h"
#include "ReconstructorCUDA.cuh"
#include "fileReaderCUDA.cuh"
namespace SLS
{

ReconstructorCUDA :: ReconstructorCUDA(const size_t projX, const size_t projY): 
    Reconstructor()
{
    projector_ = new Projector(projX, projY);
}
ReconstructorCUDA::~ReconstructorCUDA(){
    for (auto &cam: cameras_)
        delete cam;
    delete projector_;
}
void ReconstructorCUDA::addCamera(Camera *cam)
{
    cameras_.push_back(cam);
}
void ReconstructorCUDA::renconstruct()
{
    // For each camera
    for(size_t camIdx = 0; camIdx < cameras_.size(); camIdx++)
    {
        auto &cam = cameras_[camIdx];
        LOG::writeLog("Generating reconstruction bucket for \"%s\" ... \n", cam->getName().c_str());
        cam->computeShadowsAndThreasholds();    // can it be done in GPU?
        size_t x=0,y=0,xTimesY=0;
        cam->getResolution(x,y);
        xTimesY=x*y;
        cam->nextFrame();cam->nextFrame();//skip first two frames
        // Load all images into GPU memory
        uchar *images_d=nullptr;
        gpuErrchk(hipMalloc((void**)&images_d, sizeof(uchar)*xTimesY*projector_->getRequiredNumFrames()*2));
        Dynamic_Bitset_Array bitsetArray(xTimesY, projector_->getRequiredNumFrames());

        //Skip first two frames;
        cam->getNextFrame(); cam->getNextFrame();
        //Preparing data
        for (size_t i=0; i<projector_->getRequiredNumFrames(); i++)
        {
            auto frm = cam->getNextFrame();
            auto invFrm = cam->getNextFrame();
            assert(frm.isContinuous() && invFrm.isContinuous());
            gpuErrchk( hipMemcpy( &images_d[xTimesY*2*i], frm.data, 
                    sizeof(uchar)*xTimesY, hipMemcpyHostToDevice));
            gpuErrchk( hipMemcpy( &images_d[xTimesY*(2*i+1)], invFrm.data, 
                    sizeof(uchar)*xTimesY, hipMemcpyHostToDevice));
        }
        FileReaderCUDA *cudaCam = dynamic_cast<FileReaderCUDA*> (cam);
        assert(cam != nullptr);
        //buildBucket_kernel<<<200, 200>>> 
        buildBucket_kernel<<<200,200>>>
        (
                images_d, 
                projector_->getRequiredNumFrames(),
                xTimesY,
                cam->getWhiteThreshold(),
                cudaCam->getMask()->getGPUOBJ(),
                bitsetArray.getGPUOBJ()
                );
        //Check for errors
        gpuErrchk(hipPeekAtLastError());

        bitsetArray.writeToPGM("patter"+cam->getName()+".pgm", x, y, false,4800);

        //uint *patternDec_d;
        //gpuErrchk( hipMalloc((void**)&patternDec_d, sizeof(uint)*xTimesY));
        //gpuErrchk( hipMemset(patternDec_d, 200, sizeof(uint)*xTimesY));

        //bucket2uint_kernel<<<200,200>>> (
        //        bitsetArray.getGPUOBJ(),
        //        xTimesY,
        //        patternDec_d);

        //gpuErrchk(hipPeekAtLastError());

        // debugging the pattern by write the to file


        //uint *patternDec_h = new uint[xTimesY];
        //printf("Device: %p, Host: %p and sizeof uint is %d\n", patternDec_d, patternDec_h, sizeof(uint));

        //gpuErrchk( hipMemcpy(patternDec_h, patternDec_d, sizeof(uint)*xTimesY, hipMemcpyDeviceToHost));


        //assert( uint2PGM( "test"+cam->getName()+".pgm", x, y, patternDec_h,(uint)1048576 ));

        //delete[] patternDec_h;
        //gpuErrchk(hipFree(patternDec_d));

        gpuErrchk(hipFree(images_d));
    }
}

// Kernels 
//
__global__ void testBitset_kernel(
        const uchar * imgs,
        size_t numImgs,
        size_t XtimesY,
        uchar whiteThreshold,
        Dynamic_Bitset_Array_GPU mask,
        Dynamic_Bitset_Array_GPU patterns
        )
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < XtimesY)
    {
        //
        patterns.setBit(idx%40, idx);
        idx += stride;
    }
}

__global__ void buildBucket_kernel(
        const uchar * imgs,
        size_t numImgs,
        size_t XtimesY,
        uchar whiteThreshold,
        Dynamic_Bitset_Array_GPU mask,
        Dynamic_Bitset_Array_GPU patterns
        )
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    const size_t BITS_PER_BYTE = mask.BITS_PER_BYTE;
    while (idx < XtimesY)   // For each pixel
    {
        for (size_t i = 0; i<numImgs; i++)
        {
            if (!mask.getBit(0, idx)) continue;
            uchar pixel = imgs[ idx + XtimesY*(2*i)];
            uchar invPixel = imgs[ idx + XtimesY*(2*i+1)];
            if (invPixel > pixel && invPixel-pixel >= whiteThreshold)
                patterns.clearBit(i, idx);
            else if (pixel > invPixel && pixel-invPixel > whiteThreshold)
                patterns.setBit(i, idx);
            else
                mask.clearBit(0, idx);
        }
        idx += stride;
    }
}

__global__ void bucket2uint_kernel(
        Dynamic_Bitset_Array_GPU patterns,
        size_t XtimesY,
        uint * output)
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < XtimesY)
    {
        output[idx] = patterns.to_uint(idx);
        idx += stride;
    }
}

} // namespace SLS
