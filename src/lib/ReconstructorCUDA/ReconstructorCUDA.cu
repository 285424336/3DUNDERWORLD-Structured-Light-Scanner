#include "hip/hip_runtime.h"
#include "ReconstructorCUDA.cuh"
#include "fileReaderCUDA.cuh"
namespace SLS
{

ReconstructorCUDA :: ReconstructorCUDA(const size_t projX, const size_t projY): 
    Reconstructor()
{
    projector_ = new Projector(projX, projY);
}
ReconstructorCUDA::~ReconstructorCUDA(){
    for (auto &cam: cameras_)
        delete cam;
    delete projector_;
}
void ReconstructorCUDA::addCamera(Camera *cam)
{
    cameras_.push_back(cam);
}
void ReconstructorCUDA::renconstruct()
{
    // For each camera, hack
    GPUBuckets buckets[2] =
    {
        GPUBuckets( (1<<20)-1,21),
        GPUBuckets( (1<<20)-1,21)
    };
    
    /**** Profile *****/
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    /**/


    for(size_t camIdx = 0; camIdx < cameras_.size(); camIdx++)
    {
        auto &cam = cameras_[camIdx];
        LOG::writeLog("Generating reconstruction bucket for \"%s\" ... \n", cam->getName().c_str());
        cam->computeShadowsAndThreasholds();    // can it be done in GPU?
        size_t x=0,y=0,xTimesY=0;
        cam->getResolution(x,y);
        xTimesY=x*y;
        cam->nextFrame();cam->nextFrame();//skip first two frames
        // Load all images into GPU memory
        uchar *images_d=nullptr;
        gpuErrchk(hipMalloc((void**)&images_d, sizeof(uchar)*xTimesY*projector_->getRequiredNumFrames()*2));
        Dynamic_Bitset_Array bitsetArray(xTimesY, projector_->getRequiredNumFrames());

        //Skip first two frames;
        cam->getNextFrame(); cam->getNextFrame();
        //Preparing data
        for (size_t i=0; i<projector_->getRequiredNumFrames(); i++)
        {
            auto frm = cam->getNextFrame();
            auto invFrm = cam->getNextFrame();
            assert(frm.isContinuous() && invFrm.isContinuous());
            gpuErrchk( hipMemcpy( &images_d[xTimesY*2*i], frm.data, 
                    sizeof(uchar)*xTimesY, hipMemcpyHostToDevice));
            gpuErrchk( hipMemcpy( &images_d[xTimesY*(2*i+1)], invFrm.data, 
                    sizeof(uchar)*xTimesY, hipMemcpyHostToDevice));
        }
        FileReaderCUDA *cudaCam = dynamic_cast<FileReaderCUDA*> (cam);
        assert(cam != nullptr);

        Kernel::genPatternArray<<<200,200>>> (
                images_d, 
                projector_->getRequiredNumFrames(),
                xTimesY,
                cam->getWhiteThreshold(),
                cudaCam->getMask()->getGPUOBJ(),
                bitsetArray.getGPUOBJ()
                );
        //Check for errors
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipFree(images_d)); // Release the heavy image array

        Kernel::buildBuckets<<<200, 200>>> (
             cudaCam->getMask()->getGPUOBJ(),
             bitsetArray.getGPUOBJ(),
             xTimesY,
             buckets[camIdx].getGPUOBJ()
            );
        gpuErrchk(hipPeekAtLastError());


        //bitsetArray.writeToPPM("patter16"+cam->getName()+".pgm", x, y, false,(1<<16)-1);
        //bitsetArray.writeToPPM("patter17"+cam->getName()+".pgm", x, y, false,(1<<17)-1);
        //bitsetArray.writeToPPM("patter18"+cam->getName()+".pgm", x, y, false,(1<<18)-1);
        //bitsetArray.writeToPPM("patter19"+cam->getName()+".pgm", x, y, false,(1<<19)-1);
        //bitsetArray.writeToPPM("patter20"+cam->getName()+".ppm", x, y, false,(1<<20)-1);
    }


    // A lot of hacks down there, need to be refactored
    auto camera0 = dynamic_cast<FileReaderCUDA*>(cameras_[0]);
    auto camera1 = dynamic_cast<FileReaderCUDA*>(cameras_[1]);

    float* cloud = nullptr;

    gpuErrchk ( hipMalloc((void**)&cloud, buckets[0].getNumBKTs()*sizeof(float)*4));

    // Reconstructing point cloud
    LOG::writeLog("Reconstructing point cloud ...\n");
    Kernel::getPointCloud2Cam<<<200,200>>>(
            buckets[0].getGPUOBJ(),
            camera0->getMask()->getGPUOBJ(),
            camera0->getDeviceCamMat(),
            camera0->getDeviceDistMat(),
            camera0->getDeviceCamTransMat(),

            buckets[1].getGPUOBJ(),
            camera1->getMask()->getGPUOBJ(),
            camera1->getDeviceCamMat(),
            camera1->getDeviceDistMat(),
            camera1->getDeviceCamTransMat(),

            4896,3264,

            cloud
            );
    gpuErrchk( hipPeekAtLastError());
    pointCloud_.resize(buckets[0].getNumBKTs()*4);
    gpuErrchk( 
            hipMemcpy(  &pointCloud_[0],cloud, buckets[0].getNumBKTs()*sizeof(float)*4, hipMemcpyDeviceToHost));
    /**** Profile *****/
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    LOG::writeLog("GPU Time : %fms\n", milliseconds);
    /*****/
    gpuErrchk( hipFree(cloud));
    LOG::writeLog("Done\n");
}

namespace Kernel{
// Kernels 

__global__ void genPatternArray(
        const uchar * imgs,
        size_t numImgs,
        size_t XtimesY,
        uchar whiteThreshold,
        Dynamic_Bitset_Array_GPU mask,
        Dynamic_Bitset_Array_GPU patterns
        )
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < XtimesY)   // For each pixel
    {
        for (size_t i = 0; i<numImgs; i++)
        {
            if (!mask.getBit(0, idx)) 
            {
                // set the bit to black, unnecessary
                for (size_t j=0; j<numImgs; j++)
                    patterns.clearBit(j, idx);
                continue;
            }
            uchar pixel = imgs[ idx + XtimesY*(2*i)];
            uchar invPixel = imgs[ idx + XtimesY*(2*i+1)];
            if (invPixel > pixel && invPixel-pixel >= whiteThreshold)
                patterns.clearBit(i, idx);
            else if (pixel > invPixel && pixel-invPixel > whiteThreshold)
                patterns.setBit(i, idx);
            else
                mask.clearBit(0, idx);
        }
        idx += stride;
    }
}


__global__ void buildBuckets(
        Dynamic_Bitset_Array_GPU mask,
        Dynamic_Bitset_Array_GPU patterns,
        size_t XtimesY,

        GPUBucketsObj bkts
        )
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < XtimesY)   // For each pixel
    {
        if (mask.getBit(0, idx))
            bkts.add2Bucket(idx, patterns.to_uint(idx));
        idx += stride;
    }
}

__global__ void getPointCloud2Cam(
        GPUBucketsObj buckets0,
        Dynamic_Bitset_Array_GPU mask0,
        float *camMat0,
        float *distMat0,
        float *camTransMat0,

        GPUBucketsObj buckets1,
        Dynamic_Bitset_Array_GPU mask1,
        float *camMat1,
        float *distMat1,
        float *camTransMat1,


        uint camResX,
        uint camResY,

        float* pointCloud
        )
{
    //Each thread takes care of one projector pixel
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    while (idx < buckets0.NUM_BKTS_)   // For each pixel
    {
        if ( buckets0.count_[idx] == 0 || buckets1.count_[idx] == 0) 
        {
            memset( &pointCloud[4*idx], 0, sizeof(float)*4);
        }
        else
        {
            //Undistorted pixels
            float minDist = 99999.0;
            float minMidPoint[4];

            for (uint i=0; i<buckets0.count_[idx]; i++)
                for (uint j=0; j<buckets1.count_[idx]; j++)
                {

                    float undistorted0[2];
                    float undistorted1[2];

                    //Pick the first pixel in both buckets to test
                    undistortPixel(
                            buckets0.data_[idx*buckets0.MAX_CNT_PER_BKT_+i],
                            camResX, camResY,
                            camMat0, distMat0,
                            undistorted0);
                    undistortPixel(
                            buckets1.data_[idx*buckets1.MAX_CNT_PER_BKT_+j],
                            camResX, camResY,
                            camMat1, distMat1,
                            undistorted1);

                    float origin0[4];
                    float origin1[4];
                    float dir0[4];
                    float dir1[4];

                    getRay(undistorted0, camMat0, camTransMat0, 
                            origin0, dir0);
                    getRay(undistorted1, camMat1, camTransMat1, 
                            origin1, dir1);

                    float midPoint[4];

                    auto dist = getMidPoint(
                            origin0, dir0, origin1, dir1,
                            midPoint);
                    if (dist < minDist)
                    {
                        minDist = dist;
                        memcpy (minMidPoint, midPoint, sizeof(float)*4);
                    }
                }
            if (minDist < 0.3)
                memcpy ( &pointCloud[4*idx], minMidPoint, sizeof(float)*4);
            else
                memset( &pointCloud[4*idx], 0, sizeof(float)*4);

        }
        idx += stride;
    }
}

} // namespace Kernel
} // namespace SLS
