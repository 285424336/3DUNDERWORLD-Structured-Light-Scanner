#include "hip/hip_runtime.h"

#include "fileReaderCUDA.cuh"
#include "CUDA_Error.cuh"

namespace SLS
{
__global__ void computeMask_kernel(
        unsigned char *brightImg,
        unsigned char *darkImg,
        uchar blackThreashold,
        size_t resX,
        size_t resY,
        Dynamic_Bitset_Array_GPU mask
        )
{
    uint idx = blockIdx.x*blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    //const size_t BITS_PER_BYTE = mask.BITS_PER_BYTE;
    while (idx < resX*resY)
    {
        uchar b = brightImg[idx];
        uchar d = darkImg[idx];
        if (b - d > blackThreashold)
            mask.setBit(0,idx);
        else
            mask.clearBit(0,idx);
        idx += stride;
    }
}
void FileReaderCUDA::computeShadowsAndThreasholds()
{
    cv::Mat& brightImg=images_[0];
    cv::Mat& darkImg=images_[1];
    unsigned char *brightImg_d=nullptr;
    unsigned char *darkImg_d=nullptr;

    gpuErrchk( hipMalloc( (void**)&brightImg_d, sizeof(uchar)*brightImg.cols*brightImg.rows));
    gpuErrchk( hipMalloc( (void**)&darkImg_d, sizeof(uchar)*darkImg.cols*darkImg.rows));
    //Check if continous
    if (brightImg.isContinuous() && darkImg.isContinuous())
    {
        gpuErrchk(hipMemcpy( brightImg_d, brightImg.data, sizeof(uchar)*brightImg.rows*brightImg.cols, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy( darkImg_d, darkImg.data, sizeof(uchar)*darkImg.rows*darkImg.cols, hipMemcpyHostToDevice));
    }
    else
    {
        LOG::writeLogErr("images are not continuous!\n");
        exit(0);
    }

    // Intialize maskGPU_
    maskGPU_ = new Dynamic_Bitset_Array(brightImg.rows*brightImg.cols, 1);
    computeMask_kernel<<<200,200>>> (
            brightImg_d, darkImg_d, 
            blackThreshold_,
            resX_, resY_,
            maskGPU_->getGPUOBJ());
    gpuErrchk( hipPeekAtLastError());
    maskGPU_->writeToPGM(name_+"_mask.pgm", resX_, resY_, false, 1);

    // Clean up
    gpuErrchk(hipFree(brightImg_d));
    gpuErrchk(hipFree(darkImg_d));
}
} // namespace SLS
